#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <string>

#define BLOCK_SIZE 16

// Kernel for rotating the image
__global__ void rotateImageKernel(const unsigned char* input, unsigned char* output,
    int width, int height, float cos_theta, float sin_theta,
    int centerX, int centerY)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        // Compute the rotated coordinates relative to center
        int srcX = static_cast<int>((x - centerX) * cos_theta + (y - centerY) * sin_theta + centerX);
        int srcY = static_cast<int>((y - centerY) * cos_theta - (x - centerX) * sin_theta + centerY);

        // Check if source coordinates are within bounds
        if (srcX >= 0 && srcX < width && srcY >= 0 && srcY < height)
        {
            output[y * width + x] = input[srcY * width + srcX];
        }
        else
        {
            output[y * width + x] = 0; // Set out-of-bound pixels to black
        }
    }
}

// Function to invoke kernel
void rotateImage(const unsigned char* input, unsigned char* output, int width, int height, float theta)
{
    unsigned char* d_input, * d_output;
    size_t imageSize = width * height * sizeof(unsigned char);

    // Allocate device memory
    hipMalloc(&d_input, imageSize);
    hipMalloc(&d_output, imageSize);

    // Copy input data to device
    hipMemcpy(d_input, input, imageSize, hipMemcpyHostToDevice);

    // Calculate rotation matrix values
    float radians = theta * M_PI / 180.0;
    float cos_theta = cosf(radians);
    float sin_theta = sinf(radians);

    int centerX = width / 2;
    int centerY = height / 2;

    // Configure kernel launch parameters
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch kernel
    rotateImageKernel << <gridDim, blockDim >> > (d_input, d_output, width, height, cos_theta, sin_theta, centerX, centerY);

    // Copy result back to host
    hipMemcpy(output, d_output, imageSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

// Function to read PGM file
bool readPGM(const std::string& filename, unsigned char*& data, int& width, int& height)
{
    std::ifstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open input file." << std::endl;
        return false;
    }

    std::string magicNumber;
    file >> magicNumber;
    if (magicNumber != "P5") {
        std::cerr << "Error: Unsupported file format." << std::endl;
        return false;
    }

    file >> width >> height;
    int maxVal;
    file >> maxVal;
    file.ignore(1); // Ignore the newline character

    // Debugging: Log dimensions and maxVal
    std::cout << "Width: " << width << ", Height: " << height << ", Max Value: " << maxVal << std::endl;

    size_t imageSize = width * height;
    data = new unsigned char[imageSize];
    file.read(reinterpret_cast<char*>(data), imageSize);

#if 0
    if (file.gcount() != imageSize) {
		printf("file.gcount() = %d, imageSize = %d\n", file.gcount(), imageSize);
        std::cerr << "Error: Unexpected end of file while reading pixel data." << std::endl;
        delete[] data;
        return false;
    }
#endif


    file.close();
    return true;
}

// Function to write PGM file
void writePGM(const std::string& filename, const unsigned char* data, int width, int height)
{
    std::ofstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open output file." << std::endl;
        return;
    }

    // Write PGM header
    file << "P5\n" << width << " " << height << "\n255\n";
    file.write(reinterpret_cast<const char*>(data), width * height);

    file.close();
}

int main()
{
    std::string inputFile = "lena.pgm";
    std::string outputFile = "rotated_output.pgm";

    unsigned char* inputImage = nullptr;
    int width, height;

    if (!readPGM(inputFile, inputImage, width, height)) {
        return -1;
    }

	printf("width = %d, height = %d\n", width, height);

    float theta = 45.0; // Rotation angle in degrees

    unsigned char* outputImage = new unsigned char[width * height];

    // Debugging: Log output dimensions
    std::cout << "Output Dimensions: " << width << "x" << height << std::endl;

    // Rotate the image
    //rotateImage(inputImage, outputImage, width, height, theta);

    // Save the rotated image
   // writePGM(outputFile, outputImage, width, height);
    writePGM(outputFile, inputImage, width, height);

    std::cout << "Image rotation completed and saved as " << outputFile << "!" << std::endl;

    // Cleanup
    delete[] inputImage;
    delete[] outputImage;

    return 0;
}
